#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <stdlib.h>

#define N 1<<10
#define BLOCK_SIZE 1<<7

__global__
void multiply(int* A, int* B, int* C)
{
    int result = 0;

    size_t i, j;

    __shared__ int sA[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ int sB[BLOCK_SIZE][BLOCK_SIZE];

    // Column of matrix A
    size_t column = blockIdx.x * blockDim.x + threadIdx.x;
    // Row of matrix B
    size_t row = blockIdx.y * blockDim.y + threadIdx.y;

    for (size_t tile = 0; tile < gridDim.x; ++tile) {
        // Column of matrix B
        i = tile * BLOCK_SIZE + threadIdx.y;
        // Row of matrix A
        j = tile * BLOCK_SIZE + threadIdx.x;

        sA[threadIdx.y][threadIdx.x] = A[row * N + j];
        sB[threadIdx.y][threadIdx.x] = B[i * N + column];

        __syncthreads();

        for (size_t k = 0; k < BLOCK_SIZE; ++k) {
            temp += sA[threadIdx.y][k] * sB[k][threadIdx.x];
        }

        __syncthreads();
    }

    C[row * N + column] = result;
}

int main(void)
{
    srand(time(NULL));

    size_t bytes;

    int* A = (int*)malloc(bytes);
    int* B = (int*)malloc(bytes);
    int* C = (int*)malloc(bytes);

    int* dA, *dB, *dC;

    hipMalloc(&dA, bytes);
    hipMalloc(&dB, bytes);
    hipMalloc(&dC, bytes);

    for (size_t i = 0; i < N; ++i) {
        for (size_t j = 0; j < N; ++i) {
            A[i * N + j] = rand() % 1<<10;
            B[i * N + j] = rand() % 1<<10;
        }
    }

    hipMemcpy(dA, A, bytes, hipMemcpyHostToDevice);
    hipMemcpy(dB, B, bytes, hipMemcpyHostToDevice);

    dim3 threads_num(BLOCK_SIZE, BLOCK_SIZE);
    dim3 blocks_num(ceil(float(N) / BLOCK_SIZE), ceil(float(M) / BLOCK_SIZE), 1);

    multiply<<<blocks_num, threads_num>>>(dA, dB, dC);

    hipError_t error_sync = hipGetLastError();
    hipError_t error_async = hipDeviceSynchronize();

    if (error_sync != hipSuccess) {
        printf(stderr, "Error in configuration: %s", hipGetErrorString(error_sync));
        goto clean;
    }

    if (error_async != hipSuccess) {
        printf(stderr, "Error on GPU: %s", hipGetErrorString(error_async));
        goto clean;
    }

    hipMemcpy(C, dC, bytes, hipMemcpyDeviceToHost);

    for (size_t i = 0; i < N; ++i) {
        for (size_t j = 0; j < N; ++j) {
            printf("%i\t", C[i * N + j]);
        }
        printf("\n");
    }

clean:
    free(A);
    free(B);
    free(C);

    hipFree(dA);
    hipFree(dB);
    hipFree(dC);

    return 0;
}
