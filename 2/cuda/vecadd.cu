#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

#define N 1<<20
#define BLOCK_SIZE 1<<7

__global__
void add(int* a, int* b, int* c) 
{
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    size_t stride = blockDim.x * gridDim.x;

    for (size_t i = index; i < N; i += stride) {
        c[i] = a[i] + b[i];
    }
}

int main(void)
{
    int *a, *b, *c;

    hipMallocManaged(&a, sizeof(int) * N);
    hipMallocManaged(&b, sizeof(int) * N);
    hipMallocManaged(&c, sizeof(int) * N);

    for (size_t i = 0; i < N; ++i) {
        a[i] = -i;
        b[i] = i * i;
    }

    size_t threads_num = BLOCK_SIZE;
    // May overflow
    size_t blocks_num = (N + threads_num - 1) / threads_num;

    add<<<blocks_num, threads_num>>>(a, b, c);

    hipDeviceSynchronize();

    for (size_t i = 0; i < N; ++i) {
        printf("%d + %d = %d\n", a[i], b[i], c[i]);
    }

    hipFree(a);
    hipFree(b);
    hipFree(c);

    return 0;
}
