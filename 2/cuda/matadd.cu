#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <stdlib.h>

#define M 1<<6
#define N 1<<8
#define BLOCK_SIZE 1<<7

__global__
void add(int* A, int* B, int* C) 
{
    size_t column = blockDim.x * blockIdx.x + threadIdx.x;
    size_t row = blockDim.y * blockIdx.y + threadIdx.y;

    if (row < M && column < N) {
        size_t tid = row * N + column;
        C[tid] = A[tid] + B[tid];
    }
}

int main(void)
{
    srand(time(NULL));

    size_t bytes = sizeof(int) * M * N;

    int* A = (int*)malloc(bytes);
    int* B = (int*)malloc(bytes);
    int* C = (int*)malloc(bytes);

    int* dA, *dB, *dC;

    hipMalloc(&dA, bytes);
    hipMalloc(&dB, bytes);
    hipMalloc(&dC, bytes);

    for (size_t i = 0; i < M; ++i) {
        for (size_t j = 0; j < N; ++j) {
            A[i * N + j] = rand() % 1<<10;
            B[i * N + j] = rand() % 1<<10;
        }
    }

    hipMemcpy(dA, A, bytes, hipMemcpyHostToDevice);
    hipMemcpy(dB, B, bytes, hipMemcpyHostToDevice);

    // Number of CUDA threads per grid block
    dim3 threads_num(BLOCK_SIZE, BLOCK_SIZE, 1);
    // Number of blocks in grid
    dim3 blocks_num(ceil(float(N) / BLOCK_SIZE), ceil(float(M) / BLOCK_SIZE), 1);

    // Apply addition
    add<<<blocks_num, threads_num>>>(dA, dB, dC);

    hipError_t error_sync = hipGetLastError();
    hipError_t error_async = hipDeviceSynchronize();

    if (error_sync != hipSuccess) {
        printf(stderr, "Error in configuration: %s", hipGetErrorString(error_sync));
        goto clean;
    }

    if (error_async != hipSuccess) {
        printf(stderr, "Error on GPU: %s", hipGetErrorString(error_async));
        goto clean;
    }

    hipMemcpy(C, dC, bytes, hipMemcpyDeviceToHost);

    for (size_t i = 0; i < M; ++i) {
        for (size_t j = 0; j < N; ++j) {
            printf("%i\t", C[i * N + j]);
        }
        printf("\n");
    }

clean:
    free(A);
    free(B);
    free(C);

    hipFree(dA);
    hipFree(dB);
    hipFree(dC);

    return 0;
}
